#include "hip/hip_runtime.h"
/* Copyright 2018 NVIDIA Corporation. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// # RTC Kernel:
// ** Volume Isosurface Raycaster **

// # Summary:
// Compute the intersection of an isosurface with a user-controlled iso value along the current
// ray segment and shade using a fixed Blinn-Phong model.

// Define the user-defined data structure
struct Isosurface_params
{
  // common lighting params
  int light_mode;  // 0=headlight, 1=orbital
  float angle;     // 0.0 angle
  float elevation; // 0.0 elevation

  float iso_min;   // 0.5, iso value in %
  float iso_max;   // 0.5, iso value in %
  int fill_up;     // 1
  int use_shading; // 1, use local phong-blinn model
  float min_alpha; // 0.05, finite difference

  float spec_fac;  // 1.0f, specular level (phong)
  float shininess; // 50.0f, shininess parameter (phong)
  float amb_fac;   // 0.2f, ambient factor
  float diff_exp;  // 2.0f, diffuse falloff (like edge enhance)

  int show_grid;  // 0, show normal grid
  int ng_num;     // 16
  float ng_width; // 0.01f

  float3 spec_color; // make_float3(1.0f), specular color

  float2 dummy; // for memory 16 bytes alignment
};

NV_IDX_VOLUME_SAMPLE_PROGRAM_PREFIX
class NV_IDX_volume_sample_program
{
  NV_IDX_VOLUME_SAMPLE_PROGRAM

private:
  const float3 ray_dir = state.m_ray_direction;

public:
  const Isosurface_params* m_isosurface_params; // define variables to bind user-defined buffer to

public:
  NV_IDX_DEVICE_INLINE_MEMBER
  void init_instance()
  {
    // Bind the contents of the buffer slot 0 to the variable
    m_isosurface_params = NV_IDX_bind_parameter_buffer<Isosurface_params>(0);
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  int execute(const NV_IDX_sample_info_self& sample_info, float4& output_color)
  {
    const NV_IDX_volume& volume = state.self;
    const float3& sample_position = sample_info.sample_position;
    const NV_IDX_colormap& colormap = volume.get_colormap();

    // retrieve parameter buffer contents (fixed values in code definition)
    float iso_min = (1.0f - m_isosurface_params->iso_min) * colormap.get_domain_min() +
      m_isosurface_params->iso_min * colormap.get_domain_max();
    float iso_max = (1.0f - m_isosurface_params->iso_max) * colormap.get_domain_min() +
      m_isosurface_params->iso_max * colormap.get_domain_max();

    if (iso_max > iso_min)
    {
      const float iso_tmp = iso_min;
      iso_min = iso_max;
      iso_max = iso_tmp;
    }

    // sample volume and colormap
    const float volume_sample = volume.sample<float>(sample_position);

    // get spatial sample points for each dimensions
    const float rh = state.m_ray_stepsize_min; // ray sampling difference
    const float vs_dr_p = volume.sample<float>(sample_position + ray_dir * rh);
    const float vs_dr_n = volume.sample<float>(sample_position - ray_dir * rh);

    // sum up threshold exceeds in both directions for iso_min
    float sum_over = float((vs_dr_p > iso_min) + (vs_dr_n > iso_min) + (volume_sample > iso_min));
    float sum_under = float((vs_dr_p < iso_min) + (vs_dr_n < iso_min) + (volume_sample < iso_min));

    // check for iso_min intersections
    if (sum_over > 0 && sum_under > 0)
    {
      // sample color
      const float4 sample_color = colormap.lookup(iso_min); // use isovalue color
      const float3 iso_normal =
        -normalize(volume.get_gradient(sample_position)); // get isosurface normal

      // check if to skip sample
      if (sample_color.w < m_isosurface_params->min_alpha)
      {
        return NV_IDX_PROG_DISCARD_SAMPLE;
      }
      else
      {
        // show normal grid lines
        if (m_isosurface_params->show_grid)
        {
          // compute angles
          const float ang_ng = M_PI / (float)(m_isosurface_params->ng_num + 1);

          // compute modulo difference
          const float ang_mx = fmodf(acos(iso_normal.x), ang_ng);
          const float ang_my = fmodf(acos(iso_normal.y), ang_ng);
          const float ang_mz = fmodf(acos(iso_normal.z), ang_ng);

          if (fabsf(ang_mx) < m_isosurface_params->ng_width)
          {
            output_color = make_float4(0.5f, 0.f, 0.f, 1.f);
            return NV_IDX_PROG_OK;
          }
          else if (fabsf(ang_my) < m_isosurface_params->ng_width)
          {
            output_color = make_float4(0.f, 0.5f, 0.f, 1.f);
            return NV_IDX_PROG_OK;
          }
          else if (fabsf(ang_mz) < m_isosurface_params->ng_width)
          {
            output_color = make_float4(0.f, 0.f, 0.5f, 1.f);
            return NV_IDX_PROG_OK;
          }
        }

        // valid intersection found
        if (m_isosurface_params->use_shading)
        {
          output_color = blinn_shader(iso_normal, sample_color);
        }
        else
        {
          // use sample color
          output_color = sample_color;
          output_color.w = 1.0f;
        }

        return NV_IDX_PROG_OK;
      }
    }
    else if ((m_isosurface_params->fill_up == 1 && sum_over >= 3) ||
      (m_isosurface_params->fill_up == -1 && sum_under >= 3))
    {
      // use iso_min color
      output_color = colormap.lookup(iso_min);
      output_color.w = 1.0f;

      return NV_IDX_PROG_OK;
    }
    else if ((m_isosurface_params->fill_up == 2 && sum_over >= 3) ||
      (m_isosurface_params->fill_up == -2 && sum_under >= 3))
    {
      // use sample color
      output_color = colormap.lookup(volume_sample);

      return NV_IDX_PROG_OK;
    }
    else
    {
      // no isosurface intersection
      // return NV_IDX_PROG_DISCARD_SAMPLE;
    }

    // sum up threshold exceeds in both directions for iso_max
    sum_over = float((vs_dr_p > iso_max) + (vs_dr_n > iso_max) + (volume_sample > iso_max));
    sum_under = float((vs_dr_p < iso_max) + (vs_dr_n < iso_max) + (volume_sample < iso_max));

    // check for iso_max intersections
    if (sum_over > 0 && sum_under > 0)
    {
      // sample color
      const float4 sample_color = colormap.lookup(iso_max); // use isovalue color
      const float3 iso_normal =
        -normalize(volume.get_gradient(sample_position)); // get isosurface normal

      // check if to skip sample
      if (sample_color.w < m_isosurface_params->min_alpha)
        return NV_IDX_PROG_DISCARD_SAMPLE;

      // show normal grid lines
      if (m_isosurface_params->show_grid)
      {
        // compute angles
        const float ang_ng = M_PI / (float)(m_isosurface_params->ng_num + 1);

        // compute modulo difference
        const float ang_mx = fmodf(acos(iso_normal.x), ang_ng);
        const float ang_my = fmodf(acos(iso_normal.y), ang_ng);
        const float ang_mz = fmodf(acos(iso_normal.z), ang_ng);

        if (fabsf(ang_mx) < m_isosurface_params->ng_width)
        {
          output_color = make_float4(0.5f, 0.f, 0.f, 1.f);
          return NV_IDX_PROG_OK;
        }
        else if (fabsf(ang_my) < m_isosurface_params->ng_width)
        {
          output_color = make_float4(0.f, 0.5f, 0.f, 1.f);
          return NV_IDX_PROG_OK;
        }
        else if (fabsf(ang_mz) < m_isosurface_params->ng_width)
        {
          output_color = make_float4(0.f, 0.f, 0.5f, 1.f);
          return NV_IDX_PROG_OK;
        }
      }

      // valid intersection found
      if (m_isosurface_params->use_shading)
      {
        output_color = blinn_shader(iso_normal, sample_color);
      }
      else
      {
        // use sample color
        output_color = sample_color;
        output_color.w = 1.0f;
      }

      return NV_IDX_PROG_OK;
    }
    else if ((m_isosurface_params->fill_up == 1 && sum_over >= 3) ||
      (m_isosurface_params->fill_up == -1 && sum_under >= 3))
    {
      // use iso_max color
      output_color = colormap.lookup(iso_max);
      output_color.w = 1.0f;

      return NV_IDX_PROG_OK;
    }
    else if ((m_isosurface_params->fill_up == 2 && sum_over >= 3) ||
      (m_isosurface_params->fill_up == -2 && sum_under >= 3))
    {
      // use sample color
      output_color = colormap.lookup(volume_sample);

      return NV_IDX_PROG_OK;
    }
    else
    {
      // no isosurface intersection
      return NV_IDX_PROG_DISCARD_SAMPLE;
    }
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  float4 blinn_shader(const float3& normal, const float4& sample_color)
  {
    const float3 diffuse_color = make_float3(sample_color);

    float NL, NH;
    if (m_isosurface_params->light_mode == 0)
    {
      NL = NH = fabsf(dot(ray_dir, normal));
    }
    else
    {
      const float theta = m_isosurface_params->angle;
      const float phi = m_isosurface_params->elevation;
      float3 light_dir = make_float3(sinf(phi) * cosf(theta), sinf(phi) * sinf(theta), cosf(phi));

      NL = fabsf(dot(light_dir, normal));
      float3 H = normalize(light_dir + ray_dir);
      NH = fabsf(dot(H, normal));
    }

    const float diff_amnt = powf(NL, m_isosurface_params->diff_exp);
    const float spec_amnt = powf(NH, m_isosurface_params->shininess);

    // compute final color (RGB)
    const float3 shade_color = diffuse_color * (m_isosurface_params->amb_fac + diff_amnt) +
      m_isosurface_params->spec_color * (m_isosurface_params->spec_fac * spec_amnt);

    return clamp(
      make_float4(shade_color.x, shade_color.y, shade_color.z, sample_color.w), 0.0f, 1.0f);
  }
}; // class NV_IDX_volume_sample_program
